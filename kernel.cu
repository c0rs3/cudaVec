#include "hip/hip_runtime.h"
﻿#include "kernel.cuh" 
// KERNELS

// Empty kernel call for context initialization
__global__ void KernelWarmup() {}

template<typename Ty_>
__global__ void addKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

template<typename Ty_>
__global__ void mulKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] * b[i];
	}
}

template<typename Ty_>
__global__ void divKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] / b[i];
	}
}

template<typename Ty_>
__global__ void addEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b;
	}
}

template<typename Ty_>
__global__ void mulEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] * b;
	}
}

template<typename Ty_>
__global__ void divEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] / b;
	}
}

template <typename Ty_>
__global__ void matmul_kernel(const Ty_* A, const Ty_* B, Ty_* C, unsigned int M, unsigned int K, unsigned int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M && col < N) {
		Ty_ sum = 0;
		for (int i = 0; i < K; ++i) {
			sum += A[row * K + i] * B[i * N + col];
		}
		C[row * N + col] = sum;
	}
}

__host__ void CUDAContextInit(int device = 0) {
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return;
	}

	KernelWarmup << <1, 1 >> > ();
	hipDeviceSynchronize();
}

template<typename Ty_>
std::vector<Ty_> matmul_flat(const std::vector<Ty_>& A, const std::vector<Ty_>& B, unsigned int M, unsigned int K, unsigned int N) {
	std::vector<Ty_> C(M * N, 0);

	for (unsigned int i = 0; i < M; i++) {
		for (unsigned int k = 0; k < K; k++) {
			Ty_ a_ik = A[i * K + k];
			for (unsigned int j = 0; j < N; j++) {
				C[i * N + j] += a_ik * B[k * N + j];
			}
		}
	}

	return C;
}

template<typename Ty_>
std::vector<Ty_> matmul_avx(const Ty_* A, const Ty_* B, unsigned int M, unsigned int K, unsigned int N) {
	Ty_* C = new Ty_[M * N];
	std::memset(C, 0, sizeof(Ty_) * M * N);

	for (unsigned int i = 0; i < M; ++i) {
		for (unsigned int j = 0; j < N; j += 8) {
			__m256 c_vec = _mm256_setzero_ps();

			for (unsigned int k = 0; k < K; ++k) {
				__m256 b_vec;
				if (j + 8 <= N) {
					b_vec = _mm256_loadu_ps(&B[k * N + j]);
				}
				else {
					// Tail handling
					float tmp[8] = {};
					for (unsigned int t = 0; t < N - j; ++t)
						tmp[t] = B[k * N + j + t];
					b_vec = _mm256_loadu_ps(tmp);
				}

				__m256 a_val = _mm256_set1_ps(A[i * K + k]);
				c_vec = _mm256_fmadd_ps(a_val, b_vec, c_vec);
			}

			if (j + 8 <= N) {
				_mm256_storeu_ps(&C[i * N + j], c_vec);
			}
			else {
				float tmp[8];
				_mm256_storeu_ps(tmp, c_vec);
				for (unsigned int t = 0; t < N - j; ++t)
					C[i * N + j + t] = tmp[t];
			}
		}
	}

	std::vector<Ty_> result(C, C + M * N);
	delete[] C;
	return result;
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const std::vector<Ty_>& b, KernelFunc kernelFunction) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size = a.size() > b.size() ? b.size() : a.size();

	// Pinned memory pointer
	Ty_* c = nullptr;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);


	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, size * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size * sizeof(Ty_));
	hipMalloc(&dev_b, size * sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	cudaStatus = hipMemcpyAsync(dev_b, b.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 blocksPerGrid(1024);
	dim3 threadsPerBlock(size / 1024);
	kernelFunction << <blocksPerGrid, threadsPerBlock, 0, stream >> > (c, dev_a, dev_b, size);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + size);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	return res;
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const Ty_& b, KernelFunc kernelFunction) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size = a.size();

	// Pinned memory pointer
	Ty_* c = nullptr;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, size * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size * sizeof(Ty_));
	hipMalloc(&dev_b, sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}
	cudaStatus = hipMemcpyAsync(dev_b, &b, sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 blocksPerGrid(1024);
	dim3 threadsPerBlock(size / 1024);
	kernelFunction << <blocksPerGrid, threadsPerBlock, 0, stream >> > (c, dev_a, dev_b, size);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + size);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		return {};
	}

	return res;
}

template <typename Ty_>
__host__ std::vector<Ty_> matrixMul(const std::vector<Ty_>& a, const std::vector<Ty_>& b, unsigned int M, unsigned int K, unsigned int N) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size_a = M * K;
	size_t size_b = K * N;

	// Pinned memory pointer
	Ty_* c = nullptr;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, M * N * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size_a * sizeof(Ty_));
	hipMalloc(&dev_b, size_b * sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size_a * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	cudaStatus = hipMemcpyAsync(dev_b, b.data(), size_b * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid((N + 15) / 16, (M + 15) / 16);
	matmul_kernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (dev_a, dev_b, c, M, K, N);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + M * N);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		return {};
	}

	return res;
}

void bench(size_t start_dim = 1, size_t end_dim = 16) {

	for (size_t k = start_dim; k <= end_dim; k++) {
		const size_t size = 1 << k * 2;
		const size_t dim = 1 << k;

		std::vector<float> A(size);
		std::vector<float> B(size);
		for (size_t i = 0; i < size; ++i) {
			A[i] = i;
			B[i] = i;
		}
		std::clog << "Element size:" << (1 << k * 2) << " || Dimensions: " << dim << "x" << dim << endl;
		
		{
			std::clog << "AVX:" << endl;
			benchmark::Timer<float> timer1;
			matmul_avx(A.data(), B.data(), dim, dim, dim);
		}		
		std::clog << endl;

		{
			std::clog << "CUDA:" << endl;
			benchmark::Timer<float> timer2;
			matrixMul(A, B, dim, dim, dim);
		}
		std::clog << endl;

		{
			std::clog << "CPU:" << endl;
			benchmark::Timer<float> timer4;
			matmul_flat(A, B, dim, dim, dim);
		}
		std::clog << endl;


	}
}

template <typename Ty_>
void test_matrix_multiplication_correctness(size_t dim) {
	const size_t size = dim * dim;
	std::vector<Ty_> A(size), B(size);

	for (size_t i = 0; i < size; ++i) {
		A[i] = static_cast<Ty_>(i);
		B[i] = static_cast<Ty_>(i);
	}

	// All implementation results
	std::vector<float> res_avx;
	if (std::is_same<Ty_, float>::value) {
		std::vector<float> Af(size), Bf(size);

		for (size_t i = 0; i < size; ++i) {
			Af[i] = static_cast<float>(i);
			Bf[i] = static_cast<float>(i);
		}
		res_avx = matmul_avx(Af.data(), Bf.data(), dim, dim, dim);
	}
	std::vector<Ty_> res_flat = matmul_flat(A, B, dim, dim, dim);
	std::vector<Ty_> res_cuda = matrixMul(A, B, dim, dim, dim);

	auto check_equal = [&](const std::vector<Ty_>& computed, const std::vector<Ty_>& reference, const std::string& label) {
		std::cout << "check_equal for " << label << std::endl;
		for (size_t i = 0; i < size; ++i) {
			if (i % 100 == 0)
				std::cout << "\rLines remaining: " << size - i << " " << std::flush;

			if (std::abs(computed[i] - reference[i]) > (pow(10, log10(computed[i]) - 1))) {
				std::cerr << std::endl << label << " mismatch at index " << i
					<< ": got " << computed[i]
					<< ", expected " << reference[i] << std::endl;
					assert(false);
			}
		}
		std::cout << "\rLines remaining: " << 0 << " " << std::flush << std::endl;

		};

	auto check_equal_f = [&](const std::vector<float>& computed, const std::vector<Ty_>& reference, const std::string& label) {
		std::cout << "check_equal for " << label << std::endl;
		for (size_t i = 0; i < size; ++i) {
			if (i % 100 == 0)
				std::cout << "\rLines remaining: " << size - i << " " << std::flush;

			if (std::abs(computed[i] - reference[i]) > (pow(10, log10(computed[i]) - 1))) {
				std::cerr << std::endl << label << " mismatch at index " << i
					<< ": got " << computed[i]
					<< ", expected " << reference[i] << std::endl;
					assert(false);
			}
		}
		std::cout << "\rLines remaining: " << 0 << " " << std::flush << std::endl;
		};

	if (std::is_same<Ty_, float>::value) {
		check_equal_f(res_avx, res_flat, "AVX");
	}
	check_equal(res_cuda, res_flat, "CUDA");

	std::clog << "All implementations passed correctness test for size " << dim << "x" << dim << ".\n";
}



int main() {
	CUDAContextInit();
	// test_matrix_multiplication_correctness<float>(static_cast<size_t>(1) << 10);
	bench();
	return 0;
}
