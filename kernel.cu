#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include "benchmark.h"
#include <iostream>
#include <immintrin.h>

typedef std::vector<std::vector<int>> Matrix;

using std::cout;
using std::endl;
using std::cerr;
using std::flush;


// KERNELS

__global__ static void KernelWarmup() {
}

template<typename Ty_>
__global__ static void addKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

template<typename Ty_>
__global__ static void mulKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] * b[i];
	}
}

template<typename Ty_>
__global__ static void divKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] / b[i];
	}
}

template<typename Ty_>
__global__ static void addEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b;
	}
}

template<typename Ty_>
__global__ static void mulEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] * b;
	}
}

template<typename Ty_>
__global__ static void divEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] / b;
	}
}

template <typename Ty_>
__global__ static void matmul_kernel(const Ty_* A, const Ty_* B, Ty_* C, unsigned int M, unsigned int K, unsigned int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M && col < N) {
		Ty_ sum = 0;
		for (int i = 0; i < K; ++i) {
			sum += A[row * K + i] * B[i * N + col];
		}
		C[row * N + col] = sum;
	}
}

// HOST FUNCTIONS

__host__ void CUDAContextInit(int device = 0) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(device);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return;
	}

	KernelWarmup << <1, 1 >> > ();
	hipDeviceSynchronize();
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const std::vector<Ty_>& b, KernelFunc kernelFunction);

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const Ty_& b, KernelFunc kernelFunction);

template <typename Ty_>
__host__ std::vector<Ty_> matrixMul(const std::vector<Ty_>& a, const std::vector<Ty_>& b, unsigned int M, unsigned int K, unsigned int N);

template<typename Ty_>
std::vector<Ty_> operator+(const std::vector<Ty_>& left, const std::vector<Ty_>& right) {
	return performOperator(left, right, addKernel);
}

template<typename Ty_>
std::vector<Ty_> operator+(const std::vector<Ty_>& left, const Ty_& right) {
	return performOperator(left, right, addKernel);
}

template<typename Ty_>
std::vector<Ty_> operator*(const std::vector<Ty_>& left, const std::vector<Ty_>& right) {
	return performOperator(left, right, addKernel);
}

template<typename Ty_>
std::vector<Ty_> matmul_flat(const std::vector<Ty_>& A, const std::vector<Ty_>& B, unsigned int M, unsigned int K, unsigned int N) {
	std::vector<Ty_> C(M * N, Ty_(0));

	for (unsigned int i = 0; i < M; ++i) {
		for (unsigned int k = 0; k < K; ++k) {
			Ty_ a_ik = A[i * K + k];
			for (unsigned int j = 0; j < N; ++j) {
				C[i * N + j] += a_ik * B[k * N + j];
			}
		}
	}

	return C;
}

template<typename Ty_>
std::vector<Ty_> matmul_avx(const Ty_* A, const Ty_* B, unsigned int M, unsigned int K, unsigned int N) {
	Ty_* C = new Ty_[M * N];
	for (int i = 0; i < M; ++i) {
		for (int j = 0; j < N; j += 8) { // 8 floats per AVX register
			__m256 c_vec = _mm256_setzero_ps();

			for (int k = 0; k < K; ++k) {
				__m256 b_vec = _mm256_loadu_ps(&B[k * N + j]);
				__m256 a_val = _mm256_set1_ps(A[i * K + k]);
				c_vec = _mm256_fmadd_ps(a_val, b_vec, c_vec);
			}

			_mm256_storeu_ps(&C[i * N + j], c_vec);
		}
	}
	return std::vector<Ty_>(C, C + M * N);
}

int main() {
	CUDAContextInit();
	for (size_t i = 0; i < 10; i++) {

	}
	const size_t size = 1 << 16;
	const size_t dim = 1 << 8;
	std::vector<float> A(size);
	std::vector<float> B(size);

	for (size_t i = 0; i < size; ++i) {
		A[i] = i;
		B[i] = i;
	}

	{
		std::clog << "AVX:\n";
		benchmark::Timer<float> timer;
		matmul_avx(A.data(), B.data(), dim, dim, dim);
	}
	std::chrono::duration<float> dur1 = benchmark::dur;
	// std::clog << "Duration: " << std::chrono::duration_cast<std::chrono::milliseconds>(benchmark::dur).count() << "ms" << endl;

	{
		std::clog << "CUDA:\n";
		benchmark::Timer<float> timer;
		matrixMul(A, B, dim, dim, dim);
	}
	std::chrono::duration<float> dur2 = benchmark::dur;
	if (!dur1.count() > dur2.count()) {
		std::cout << "stop\n";
		break;
	}
	// std::clog << "Duration: " << std::chrono::duration_cast<std::chrono::milliseconds>(benchmark::dur).count() << "ms" << endl;
	return 0;
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const std::vector<Ty_>& b, KernelFunc kernelFunction) {
	CUDAContextInit();
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size = a.size() > b.size() ? b.size() : a.size();

	// Pinned memory pointer
	Ty_* c;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);


	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, size * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size * sizeof(Ty_));
	hipMalloc(&dev_b, size * sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	cudaStatus = hipMemcpyAsync(dev_b, b.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 blocksPerGrid(1024);
	dim3 threadsPerBlock(size / 1024);
	kernelFunction << <blocksPerGrid, threadsPerBlock, 0, stream >> > (c, dev_a, dev_b, size);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + size);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	return res;
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const Ty_& b, KernelFunc kernelFunction) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size = a.size();

	// Pinned memory pointer
	Ty_* c;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, size * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size * sizeof(Ty_));
	hipMalloc(&dev_b, sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}
	cudaStatus = hipMemcpyAsync(dev_b, &b, sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 blocksPerGrid(1024);
	dim3 threadsPerBlock(size / 1024);
	kernelFunction << <blocksPerGrid, threadsPerBlock, 0, stream >> > (c, dev_a, dev_b, size);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + size);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		return {};
	}

	return res;
}

template <typename Ty_>
__host__ std::vector<Ty_> matrixMul(const std::vector<Ty_>& a, const std::vector<Ty_>& b, unsigned int M, unsigned int K, unsigned int N) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size_a = M * K;
	size_t size_b = K * N;

	// Pinned memory pointer
	Ty_* c;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, M * N * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size_a * sizeof(Ty_));
	hipMalloc(&dev_b, size_b * sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size_a * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	cudaStatus = hipMemcpyAsync(dev_b, b.data(), size_b * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid((N + 15) / 16, (M + 15) / 16);
	matmul_kernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (dev_a, dev_b, c, M, K, N);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + M * N);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		return {};
	}

	return res;
}