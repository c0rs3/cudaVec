#include "hip/hip_runtime.h"
﻿#include "kernel.cuh" 
// KERNELS

// empty kernel call for context initialization
__global__ void KernelWarmup() {
}

template<typename Ty_>
__global__ void addKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

template<typename Ty_>
__global__ void mulKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] * b[i];
	}
}

template<typename Ty_>
__global__ void divKernel(Ty_* c, const Ty_* a, const Ty_* b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] / b[i];
	}
}

template<typename Ty_>
__global__ void addEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b;
	}
}

template<typename Ty_>
__global__ void mulEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] * b;
	}
}

template<typename Ty_>
__global__ void divEqualsKernel(Ty_* c, const Ty_* a, const Ty_& b, unsigned int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] / b;
	}
}

template <typename Ty_>
__global__ void matmul_kernel(const Ty_* A, const Ty_* B, Ty_* C, unsigned int M, unsigned int K, unsigned int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M && col < N) {
		Ty_ sum = 0;
		for (int i = 0; i < K; ++i) {
			sum += A[row * K + i] * B[i * N + col];
		}
		C[row * N + col] = sum;
	}
}

__host__ void CUDAContextInit() {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return;
	}

	KernelWarmup << <1, 1 >> > ();
	hipDeviceSynchronize();
}

template<typename Ty_>
std::vector<Ty_> matmul_flat(const std::vector<Ty_>& A, const std::vector<Ty_>& B, unsigned int M, unsigned int K, unsigned int N) {
	std::vector<Ty_> C(M * N, 0);

	for (unsigned int i = 0; i < M; ++i) {
		for (unsigned int k = 0; k < K; ++k) {
			Ty_ a_ik = A[i * K + k];
			for (unsigned int j = 0; j < N; ++j) {
				C[i * N + j] += a_ik * B[k * N + j];
			}
		}
	}

	return C;
}

template<typename Ty_>
std::vector<Ty_> matmul_avx(const Ty_* A, const Ty_* B, unsigned int M, unsigned int K, unsigned int N) {
	Ty_* C = new Ty_[M * N];
	std::memset(C, 0, sizeof(Ty_) * M * N);

	for (unsigned int i = 0; i < M; ++i) {
		for (unsigned int j = 0; j < N; j += 8) {
			__m256 c_vec = _mm256_setzero_ps();

			for (unsigned int k = 0; k < K; ++k) {
				__m256 b_vec;
				if (j + 8 <= N) {
					b_vec = _mm256_loadu_ps(&B[k * N + j]);
				}
				else {
					// Tail handling
					float tmp[8] = {};
					for (unsigned int t = 0; t < N - j; ++t)
						tmp[t] = B[k * N + j + t];
					b_vec = _mm256_loadu_ps(tmp);
				}

				__m256 a_val = _mm256_set1_ps(A[i * K + k]);
				c_vec = _mm256_fmadd_ps(a_val, b_vec, c_vec);
			}

			if (j + 8 <= N) {
				_mm256_storeu_ps(&C[i * N + j], c_vec);
			}
			else {
				float tmp[8];
				_mm256_storeu_ps(tmp, c_vec);
				for (unsigned int t = 0; t < N - j; ++t)
					C[i * N + j + t] = tmp[t];
			}
		}
	}

	std::vector<Ty_> result(C, C + M * N);
	delete[] C;
	return result;
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const std::vector<Ty_>& b, KernelFunc kernelFunction) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size = a.size() > b.size() ? b.size() : a.size();

	// Pinned memory pointer
	Ty_* c;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);


	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, size * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size * sizeof(Ty_));
	hipMalloc(&dev_b, size * sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	cudaStatus = hipMemcpyAsync(dev_b, b.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 blocksPerGrid(1024);
	dim3 threadsPerBlock(size / 1024);
	kernelFunction << <blocksPerGrid, threadsPerBlock, 0, stream >> > (c, dev_a, dev_b, size);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + size);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	return res;
}

template <typename Ty_, typename KernelFunc>
__host__ std::vector<Ty_> performOperator(const std::vector<Ty_>& a, const Ty_& b, KernelFunc kernelFunction) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size = a.size();

	// Pinned memory pointer
	Ty_* c;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, size * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size * sizeof(Ty_));
	hipMalloc(&dev_b, sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}
	cudaStatus = hipMemcpyAsync(dev_b, &b, sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 blocksPerGrid(1024);
	dim3 threadsPerBlock(size / 1024);
	kernelFunction << <blocksPerGrid, threadsPerBlock, 0, stream >> > (c, dev_a, dev_b, size);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + size);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		return {};
	}

	return res;
}

template <typename Ty_>
__host__ std::vector<Ty_> matrixMul(const std::vector<Ty_>& a, const std::vector<Ty_>& b, unsigned int M, unsigned int K, unsigned int N) {
	// cudastatus for tracking errors
	hipError_t cudaStatus = hipSuccess;

	// Set device (GPU)
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to set device! (incompatible GPU?)" << std::endl;
		return {};
	}

	// Device pointers
	Ty_* dev_a = nullptr, * dev_b = nullptr;

	// Vector size
	size_t size_a = M * K;
	size_t size_b = K * N;

	// Pinned memory pointer
	Ty_* c;

	// CUDA stream
	hipStream_t stream;
	cudaStatus = hipStreamCreate(&stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to create stream!" << std::endl;
		hipStreamDestroy(stream);
		return {};
	}

	// Allocate pinned host memory
	hipHostMalloc(&c, M * N * sizeof(Ty_));

	// Allocate device memory
	hipMalloc(&dev_a, size_a * sizeof(Ty_));
	hipMalloc(&dev_b, size_b * sizeof(Ty_));

	// Copy data from host to device asynchronously
	cudaStatus = hipMemcpyAsync(dev_a, a.data(), size_a * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	cudaStatus = hipMemcpyAsync(dev_b, b.data(), size_b * sizeof(Ty_), hipMemcpyHostToDevice, stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed memcpy!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	// Kernel launch configuration
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid((N + 15) / 16, (M + 15) / 16);
	matmul_kernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (dev_a, dev_b, c, M, K, N);

	// Synchronize the stream to ensure all tasks are complete
	cudaStatus = hipStreamSynchronize(stream);
	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to synchronize streams!" << std::endl;
		hipFree(dev_a);
		hipFree(dev_b);
		hipHostFree(c);
		hipStreamDestroy(stream);

		return {};
	}

	std::vector<Ty_> res(c, c + M * N);

	// Cleanup
	hipFree(dev_a);
	hipFree(dev_b);
	hipHostFree(c);
	cudaStatus = hipStreamDestroy(stream);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Failed to destroy stream!" << std::endl;
		return {};
	}

	return res;
}

void bench() {
	std::vector<float> res1, res2, res3;
	for (size_t k = 1; k <= 18; k++) {
		const size_t size = 1 << k * 2;
		const size_t dim = 1 << k;
		Eigen::MatrixXd m1 = Eigen::MatrixXd::Constant(dim, dim, 2.0);
		Eigen::MatrixXd m2 = Eigen::MatrixXd::Constant(dim, dim, 2.0);

		std::vector<float> A(size);
		std::vector<float> B(size);

		for (size_t i = 0; i < size; ++i) {
			A[i] = i;
			B[i] = i;
		}
		std::clog << "Element size:" << (1 << k * 2) << endl;
		{
			std::clog << "AVX:" << endl;
			benchmark::Timer<float> timer1;
			std::vector<float> res1 = matmul_avx(A.data(), B.data(), dim, dim, dim);
		}
		auto dur1 = benchmark::dur;
		std::clog << endl;

		{
			std::clog << "CUDA:" << endl;
			benchmark::Timer<float> timer2;
			std::vector<float> res2 = matrixMul(A, B, dim, dim, dim);
		}
		auto dur2 = benchmark::dur;
		std::clog << endl;

		{
			std::clog << "Eigen:" << endl;
			benchmark::Timer<float> timer3;
			m1 = m1 * m2;
		}
		auto dur3 = benchmark::dur;
		std::clog << endl;

		{
			std::clog << "CPU:" << endl;
			benchmark::Timer<float> timer4;
			res3 = matmul_flat(A, B, dim, dim, dim);
		}
		auto dur4 = benchmark::dur;
		std::clog << endl;

	}
}

template <typename Ty_>
void test_matrix_multiplication_correctness(size_t dim) {
	const size_t size = dim * dim;
	std::vector<Ty_> A(size), B(size);

	for (size_t i = 0; i < size; ++i) {
		A[i] = static_cast<Ty_>(i);
		B[i] = static_cast<Ty_>(i);
	}

	// All implementation results
	std::vector<float> res_avx;
	if (std::is_same<Ty_, float>::value) {
		std::vector<float> Af(size), Bf(size);

		for (size_t i = 0; i < size; ++i) {
			Af[i] = static_cast<float>(i);
			Bf[i] = static_cast<float>(i);
		}
		res_avx = matmul_avx(Af.data(), Bf.data(), dim, dim, dim);
	}
	std::vector<Ty_> res_flat = matmul_flat(A, B, dim, dim, dim);
	std::vector<Ty_> res_cuda = matrixMul(A, B, dim, dim, dim);

	// Eigen is used as reference
	Eigen::Map<const Eigen::Matrix<Ty_, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>> mA(A.data(), dim, dim);
	Eigen::Map<const Eigen::Matrix<Ty_, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>> mB(B.data(), dim, dim);
	Eigen::Matrix<Ty_, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> mE = mA * mB;

	// Flatten Eigen result
	std::vector<Ty_> res_eigen(size);
	Eigen::Map<Eigen::Matrix<Ty_, Eigen::Dynamic, 1>>(res_eigen.data(), size) = Eigen::Map<Eigen::Matrix<Ty_, Eigen::Dynamic, 1>>(mE.data(), size);

	auto check_equal = [&](const std::vector<Ty_>& computed, const std::vector<Ty_>& reference, const std::string& label) {
		for (size_t i = 0; i < size; ++i) {
			if (std::abs(computed[i] - reference[i]) > 1e-3f) {
				std::cerr << label << " mismatch at index " << i
					<< ": got " << computed[i]
					<< ", expected " << reference[i] << std::endl;
					assert(false);
			}
		}
		};

	auto check_equal_f = [&](const std::vector<float>& computed, const std::vector<Ty_>& reference, const std::string& label) {
		for (size_t i = 0; i < size; ++i) {
			if (std::abs(computed[i] - reference[i]) > log10(size)) {
				std::cerr << label << " mismatch at index " << i
					<< ": got " << computed[i]
					<< ", expected " << reference[i] << std::endl;
					assert(false);
			}
		}
		};

	check_equal(res_flat, res_eigen, "Flat");
	if (std::is_same<Ty_, float>::value) {
		check_equal_f(res_avx, res_eigen, "AVX");
	}
	check_equal(res_cuda, res_eigen, "CUDA");

	std::clog << "All implementations passed correctness test for size " << dim << "x" << dim << ".\n";
}



int main() {
	CUDAContextInit();
	test_matrix_multiplication_correctness<int>(static_cast<size_t>(1) << 14);
	return 0;
}
